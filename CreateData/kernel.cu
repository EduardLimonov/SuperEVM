﻿#include <thrust/device_vector.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "datcreater.cuh"

hipError_t runGenerate();


int main()
{
    runGenerate();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t runGenerate()
{
    int maxTriang = 100, maxCircl = 200, *dev_nTriang, *dev_nCircl, resTriang = 0, resCircl = 0;
    int nBlocks = 100, nThreads = 100;
    int polySize = 10000;

    int* dev_polygon;
    int* polygon = new int[polySize * polySize];

    int *dev_a = 0;
    Point d = { polySize / nThreads, polySize / nBlocks };
    Pair<thrust::device_vector<Rect>>* conflicts;
    thrust::device_vector<Rect>* rects;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&conflicts, nBlocks * nThreads * sizeof(Pair<thrust::device_vector<Rect>>));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_polygon, polySize * polySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nTriang, 1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_nCircl, 1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&rects, 1 * sizeof(thrust::device_vector<Rect>));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    create_rects<<<nBlocks, nThreads>>>(d, maxTriang, maxCircl, conflicts, polySize, polygon, rects);

    for (int i = 0; i < 4; i++)
        // чудовищная синхронизация, но всё потому, что недоступна CUDA 9
        resolveConflicts<<<nBlocks, nThreads >>> (conflicts, i);

    create_objects<<<nBlocks, nThreads>>> (maxTriang, maxCircl, conflicts, polySize, dev_polygon, rects, dev_nTriang, dev_nCircl);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(polygon, dev_polygon, polySize * polySize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(&resTriang, dev_nTriang, 1 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(&resCircl, dev_nCircl, 1 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    std::cout << "TRIANGLES: " << resTriang << "\nCIRCLES: " << resCircl << std::endl;

Error:
    hipFree(conflicts);
    hipFree(dev_polygon);
    hipFree(dev_nTriang);
    hipFree(dev_nCircl);
    hipFree(rects);
    
    return cudaStatus;
}
