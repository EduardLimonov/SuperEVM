#include "hip/hip_runtime.h"
﻿#include "SimpleCounter.cuh"
#include "hip/hip_runtime.h"
#include ""

__device__ void run(int polySize, int* polygon, Point d, int* circles, int* triangs, int nXY)
{
	Point xy = getXY();
	Point start = Point(xy.x * d.x, xy.y * d.y);

	thrust::device_vector<int> done;

	// выполняем редукцию
	for (int iter = 1; iter < nXY; iter *= 2)
	{
		// iter -> управляем нитью, отвечающей за квадрат iter * iter сегментов
		Point end = start + d * iter;

		Point xy = getXY();
		if (xy.x % iter == 0 && xy.y % iter == 0)
		{
			for (int x = start.x; x < end.x; x++)
				for (int y = start.y; y < end.y; y++)
				{
					int idx = getByPoint(Point(x, y), polySize);
					if (
						(polygon[idx] == CIRCLE_COLOR || polygon[idx] == TRIANG_COLOR) &&
						!contains(done, idx)
						)
						if (polygon[idx] == CIRCLE_COLOR)
							*circles += make_cycle(idx, polySize, polygon, start, end, done);
						else
							*triangs += make_cycle(idx, polySize, polygon, start, end, done);
				}
		}
		__syncthreads();
	}
}

__device__ int make_cycle(int idx, int polySize, int* polygon, Point start, Point end,
	thrust::device_vector<int>& done)
{
	thrust::device_vector<int> todo;
	Point t = getByCoords(idx, polySize);
	todo.push_back(idx);
	bool got_border = false;

	while (todo.size())
	{
		int next = todo.front();
		todo.erase(todo.begin());

		done.push_back(next);
		addNeigh(todo, getByCoords(next, polySize), start, end, done, polySize, polygon);
	}

	auto start_neighs = neighs(getByCoords(done.back(), polySize), start, end, polySize, polygon);
	if (contains(start_neighs, idx))
		return 1;
	else
		return 0;
}

__device__ thrust::device_vector<int> neighs(Point pos, Point& start, Point& stop, int polySize, int* polygon)
{
	thrust::device_vector<int> res;
	int pos_idx = getByPoint(pos, polySize);

	for (int x = pos.x - 1; x <= pos.x + 1; x++)
		for (int y = pos.y - 1; y <= pos.y + 1; y++)
		{
			Point xy(x, y);
			if (xy == pos)
				continue;
			int idx = getByPoint(xy, polySize);
			if (polygon[idx] == polygon[pos_idx] && !outBorder(pos, start, stop))
				res.push_back(idx);
		}
	return res;
}

__device__ bool onBorder(Point t, Point& start, Point& end)
{
	return t.x == start.x || t.y == start.y || t.x == end.x || t.y == end.y;
}

__device__ bool outBorder(Point t, Point& start, Point& end)
{
	return t.x < start.x || t.y < start.y || t.x > end.x || t.y > end.y;
}

__device__ bool contains(thrust::device_vector<int> &v, int p)
{
	for (auto point : v)
		if (point == p)
			return true;
	return false;
}

__device__ void addNeigh(thrust::device_vector<int>& stack, Point pos, Point& start, Point& stop,
	thrust::device_vector<int>& done, int polySize, int* polygon)
{
	// Добавить в стэк еще не обойденных соседей
	int pos_idx = getByPoint(pos, polySize);

	for (int x = pos.x - 1; x <= pos.x + 1; x++)
		for (int y = pos.y - 1; y <= pos.y + 1; y++)
		{
			Point xy(x, y);
			if (xy == pos)
				continue;
			int idx = getByPoint(xy, polySize);
			if (polygon[idx] == polygon[pos_idx] && !contains(done, idx) && !outBorder(pos, start, stop))
				stack.push_back(idx);
		}
}

__device__ int _abs(int x)
{
	return x > 0 ? x : -x;
}

// ====================================================================================================

__device__ Point getXY()
{
	return Point(threadIdx.x, blockIdx.x);
}

__device__ thrust::device_vector<int>* getAwayPoint(thrust::device_vector<int>* away, Point d)
{
	return away + (threadIdx.x + d.x) + (blockIdx.x + d.y) * gridDim.x;
}

__device__ int getByPoint(Point p, int polySize)
{
	return p.y * polySize + p.x;
}

__device__ Point getByCoords(int idx, int polySize)
{
	return Point(idx % polySize, idx / polySize);
}